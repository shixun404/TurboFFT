#include "hip/hip_runtime.h"
// #include "include/turbofft/tensor.h"
#include "include/turbofft/utils.h"
#include "include/turbofft/fft/thread/fft.h"
#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hipfft/hipfftXt.h>
#define DataType float2

void test_cufft(DataType* input_d, DataType* output_d, DataType* output_cufft, size_t N){
    hipfftHandle plan;  
    hipfftCreate(&plan);
    int res = hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
    res = hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d, HIPFFT_FORWARD);
    hipMemcpy((void*)output_cufft, (void*)output_d, N * sizeof(DataType), hipMemcpyDeviceToHost);
    hipfftDestroy(plan);
}

void test_turbofft(DataType* input_d, DataType* output_d, DataType* output_turbofft, 
                    size_t N){
    dim3 gridDim(1, 1, 1); 
    dim3 blockDim(1, 1, 1);
    // turbofft::fft::thread::fft<DataType, turbofft::Tensor<DataType, 1, 2>><<<gridDim, blockDim>>>(input_d, output_d);
    // turbofft::fft::thread::fft<DataType><<<gridDim, blockDim>>>(input_d, output_d);
    turbofft::fft::thread::fft<<<gridDim, blockDim>>>(input_d, output_d);
    hipDeviceSynchronize();
    printf("%d\n",  N * sizeof(DataType));
    CUDA_CALLER(hipMemcpy((void*)output_turbofft, (void*)output_d, N * sizeof(DataType), hipMemcpyDeviceToHost));

}

void compareData(DataType* res, DataType *res_ref, size_t N, double error_bound, 
                bool printInfo=false){
    double rel_error = 0.;
    for(int i = 0; i < N; ++i){
        rel_error = abs((res[i].x - res_ref[i].x) / res_ref[i].x);
        if(rel_error > error_bound){
            printf("Error detected: res[%d].x=%f, res_ref[%d].x=%f, rel_error=%f\n", 
            i, res[i].x, i, res_ref[i].x, rel_error);
        }
        rel_error = abs((res[i].y - res_ref[i].y) / res_ref[i].y);
        if(rel_error > error_bound){
            printf("Error detected: res[%d].y=%f, res_ref[%d].y=%f, rel_error=%f\n", 
            i, res[i].y, i, res_ref[i].y, rel_error);
        }
    }
}
void printData(DataType* res, size_t N){
    double rel_error = 0.;
    for(int i = 0; i < N; ++i){
        printf("res[%d] = %f + %f j\n", i, res[i].x, res[i].y);
    }
}

void initializeData(DataType *&input, DataType *&input_d, DataType *&output_d, 
                    DataType *&output_turbofft, DataType *&output_cufft, size_t N){
    input = (DataType*)calloc(N, sizeof(DataType));
    output_turbofft = (DataType*)calloc(N, sizeof(DataType));
    output_cufft = (DataType*)calloc(N, sizeof(DataType));
    CUDA_CALLER(hipMalloc((void**)&input_d, sizeof(DataType) * N));
    CUDA_CALLER(hipMalloc((void**)&output_d, sizeof(DataType) * N));
    
    for(int i = 0; i < N; ++i){
        input[i].x = 1;
        input[i].y = 1;
    }

    CUDA_CALLER(hipMemcpy((void*)input_d, (void*)input, N * sizeof(DataType), hipMemcpyHostToDevice));
}

int main(){
    DataType* input, *output_turbofft, *output_cufft;
    DataType* input_d, *output_d;
    int N = 2;

    initializeData(input, input_d, output_d, output_turbofft, output_cufft, N);
    test_turbofft(input_d, output_d, output_turbofft, N);

    test_cufft(input_d, output_d, output_cufft, N);

    compareData(output_turbofft, output_cufft, N, 1e-5, false);
    printData(output_turbofft, N);
    printData(output_cufft, N);

    return 0;
}