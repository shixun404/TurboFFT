#include "hip/hip_runtime.h"
#include "include/turbofft.h"
// #include "include/code_gen/generated/fft_radix_2_logN_8_upload_0.cuh"

// #define DataType nv_bfloat162
#define DataType double2
// #define DataType float2
// #define DataType half2



void test_turbofft(DataType* input_d, DataType* output_d, DataType* output_turbofft, 
                    std::vector<long long int> param, long long int bs, int ntest){
    long long int N = (1 << param[0]), threadblock_bs, Ni, WorkerFFTSize;
    long long int logN = param[0];
    long long int shared_size[3], griddims[3], blockdims[3]; 
    int kernel_launch_times = param[1];
    float gflops, elapsed_time;
    hipEvent_t fft_begin, fft_end;
    printf("adasdas\n");
    for(int i = 0; i < kernel_launch_times; ++i){
        threadblock_bs = param[5 + i];
        Ni = (1 << param[2 + i]); 
        WorkerFFTSize = param[8 + i]; 
        shared_size[i] = Ni * threadblock_bs * sizeof(DataType);
        griddims[i] = N * bs / (Ni * threadblock_bs);
        blockdims[i] = (Ni * threadblock_bs) / WorkerFFTSize;
        hipFuncSetAttribute(reinterpret_cast<const void*>(turboFFTArr[logN][i]), hipFuncAttributeMaxDynamicSharedMemorySize, shared_size[i]);
    }
    
    hipEventCreate(&fft_begin);
    hipEventCreate(&fft_end);

    hipEventRecord(fft_begin);
    for (int j = 0; j < ntest; ++j){
        for(int i = 0; i < kernel_launch_times; ++i){
            turboFFTArr[logN][i]<<<griddims[i], blockdims[i], shared_size[i]>>>(input_d, output_d, bs);
        }
        hipDeviceSynchronize();
    }
    hipEventRecord(fft_end);
    hipEventSynchronize(fft_begin);
    hipEventSynchronize(fft_end);
    hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);

    
    elapsed_time = elapsed_time / ntest;
    gflops = 5 * N * log2f(N) * bs / elapsed_time * 1000 / 1000000000.f;
    
    printf("turboFFT finished: T=%8.3fms, FLOPS=%8.3fGFLOPS\n", elapsed_time, gflops);
    
    checkCudaErrors(hipMemcpy((void*)output_turbofft, (void*)output_d, N * bs * sizeof(DataType), hipMemcpyDeviceToHost));
}


int main(int argc, char *argv[]){
    if (argc != 3) {
        std::cerr << "Usage: program_name N bs" << std::endl;
        return 1;
    }

    long long logN = std::atoi(argv[1]); // Convert first argument to integer
    long long N = 1 << logN; // Convert first argument to integer
    long long bs = std::atoi(argv[2]); // Convert second argument to integer
    
    DataType* input, *output_turbofft, *output_cufft;
    DataType* input_d, *output_d;
    int ntest = 10;

    std::vector<std::vector<long long int>> params;
    
    std::ifstream file("../include/param/param.csv");
    if (file.is_open()) {
        std::cout << "File opened successfully." << std::endl;
        // Perform file operations here
    } else {
        std::cout << "Failed to open file." << std::endl;
    }    
    params = utils::load_parameters(file);


    // Verification
    utils::initializeData<DataType>(input, input_d, output_d, output_turbofft, output_cufft, N, bs + 3);
    test_turbofft(input_d, output_d, output_turbofft, params[logN], bs, 1);
    profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, N, bs, 1);
    
    utils::compareData<DataType>(output_turbofft, output_cufft, N * bs, 1e-5);

    // Profiling
    test_turbofft(input_d, output_d, output_turbofft, params[logN], bs, ntest);
    profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, N, bs, ntest);
    
    hipFree(input_d);
    hipFree(output_d);
    return 0;
}

// profiler::cufft::test_cufft_ft<DataType>(input_d, output_d, output_cufft, input_d + N * (bs + 2),
//                                          input_d + N * (bs + 1), output_d + N * (bs + 2),   N, bs, ntest);
// profiler::cufft::test_cufft_ft<DataType>(input_d, output_d, output_cufft, input_d + N * (bs + 2),
//                                          input_d + N * (bs + 1), output_d + N * (bs + 2),   N, bs, ntest);
