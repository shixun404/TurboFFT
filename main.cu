#include "hip/hip_runtime.h"
#include "include/turbofft.h"
// #define DataType nv_bfloat162
// #define DataType double2
#define DataType double2
// #define DataType half2



void test_turbofft(DataType* input_d, DataType* output_d, DataType* output_turbofft, 
                    long long int N){
    dim3 gridDim(1, 1, 1); 
    dim3 blockDim(1, 32, 1);
    // turbofft::fft::thread::fft<DataType, turbofft::Tensor<DataType, 1, 2>><<<gridDim, blockDim>>>(input_d, output_d);
    // turbofft::fft::thread::fft<DataType><<<gridDim, blockDim>>>(input_d, output_d);
    long long int shared_size = (N * sizeof(DataType) / 16) * 17;

    hipFuncSetAttribute(reinterpret_cast<const void*>(turbofft::fft::thread::fft<DataType), 8>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_size);
    turbofft::fft::thread::fft<DataType, 8><<<gridDim, blockDim, shared_size>>>(input_d, output_d);
    // turbofft::fft::thread::fft<DataType><<<gridDim, blockDim>>>(input_d, output_d);
    hipDeviceSynchronize();
    printf("%d\n",  N * sizeof(DataType));
    checkCudaErrors(hipMemcpy((void*)output_turbofft, (void*)output_d, N * sizeof(DataType), hipMemcpyDeviceToHost));

}


int main(){
    DataType* input, *output_turbofft, *output_cufft;
    DataType* input_d, *output_d;
    int N = 256, bs=1;

    utils::initializeData<DataType>(input, input_d, output_d, output_turbofft, output_cufft, N);
    test_turbofft(input_d, output_d, output_turbofft, N);

    profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, N, bs);

    utils::compareData<DataType>(output_turbofft, output_cufft, N, 1e-5);
    // printData(output_turbofft, N);
    // printData(output_cufft, N);

    return 0;
}