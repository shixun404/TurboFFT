#include "hip/hip_runtime.h"
#include "include/turbofft.h"
// #include "include/code_gen/generated/fft_radix_2_logN_8_upload_0.cuh"
#include "include/code_gen/generated/fft_radix_2_logN_2_upload_0.cuh"
// #define DataType nv_bfloat162
#define DataType double2
// #define DataType float2
// #define DataType half2



void test_turbofft(DataType* input_d, DataType* output_d, DataType* output_turbofft, 
                    std::vector<long long int> param, long long int bs, int ntest){
    long long int N = (1 << param[0]), bs, Ni, WorkerFFTSize;
    long long int shared_size[3], griddims[3], blockdims[3]; 
    int kernel_launch_times = param[1];
    float gflops, elapsed_time;
    hipEvent_t fft_begin, fft_end;
    
    for(int i = 0; i < kernel_launch_times; ++i){
        bs = param[5 + i];
        Ni = (1 << param[2 + i]); 
        WorkerFFTSize = param[8 + i]; 
        shared_size[i] = Ni * bs * sizeof(DataType);
        griddims[i] = N / (Ni * bs);
        blockdims[i] = (Ni * bs) / WorkerFFTSize;
        hipFuncSetAttribute(reinterpret_cast<const void*>(turboFFTArr[logN][i]), hipFuncAttributeMaxDynamicSharedMemorySize, shared_size[i]);
    }
    
    hipEventCreate(&fft_begin);
    hipEventCreate(&fft_end);

    hipEventRecord(fft_begin);
    for (int j = 0; j < ntest; ++j){
        for(int i = 0; i < kernel_launch_times; ++i){
            turboFFTArr[logN][i]<<<griddims[i], blockdims[i], shared_size[i]>>>(input_d, output_d);
        }
        hipDeviceSynchronize();
    }
    hipEventRecord(fft_end);
    hipEventSynchronize(fft_begin);
    hipEventSynchronize(fft_end);
    hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);

    
    elapsed_time = elapsed_time / ntest;
    gflops = 5 * N * log2f(N) * bs / elapsed_time * 1000 / 1000000000.f;
    
    printf("turboFFT finished: T=%8.3fms, FLOPS=%8.3fGFLOPS\n", elapsed_time, gflops);
    
    checkCudaErrors(hipMemcpy((void*)output_turbofft, (void*)output_d, N * sizeof(DataType), hipMemcpyDeviceToHost));
}


int main(int argc, char *argv[]){
    if (argc != 3) {
        std::cerr << "Usage: program_name N bs" << std::endl;
        return 1;
    }

    long long N = std::atoi(argv[1]); // Convert first argument to integer
    long long bs = std::atoi(argv[2]); // Convert second argument to integer
    
    DataType* input, *output_turbofft, *output_cufft;
    DataType* input_d, *output_d;
    int ntest = 10;

    std::vector<std::vector<int>> params;
    params = utils::load_parameters();


    // Verification
    utils::initializeData<DataType>(input, input_d, output_d, output_turbofft, output_cufft, N, bs + 3);
    test_turbofft(input_d, output_d, output_turbofft, params[N], bs, 1);
    profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, N, bs, 1);
    
    utils::compareData<DataType>(output_turbofft, output_cufft, N, 1e-5);

    // Profiling
    test_turbofft(input_d, output_d, output_turbofft, params[N], bs, ntest);
    profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, N, bs, ntest);
    
    hipFree(input_d);
    hipFree(output_d);
    return 0;
}

// profiler::cufft::test_cufft_ft<DataType>(input_d, output_d, output_cufft, input_d + N * (bs + 2),
//                                          input_d + N * (bs + 1), output_d + N * (bs + 2),   N, bs, ntest);
// profiler::cufft::test_cufft_ft<DataType>(input_d, output_d, output_cufft, input_d + N * (bs + 2),
//                                          input_d + N * (bs + 1), output_d + N * (bs + 2),   N, bs, ntest);
