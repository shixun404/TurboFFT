#include "hip/hip_runtime.h"
#include "include/turbofft.h"
// #include "include/code_gen/generated/fft_radix_2_logN_8_upload_0.cuh"

// #define DataType nv_bfloat162
#define DataType double2
// #define DataType float2
// #define DataType half2



void test_turbofft( DataType* input_d, DataType* output_d, DataType* output_turbofft,
                    DataType* twiddle_d, std::vector<long long int> param, 
                    long long int bs, int ntest){
    long long int N = (1 << param[0]), threadblock_bs, Ni, WorkerFFTSize;
    long long int logN = param[0];
    long long int shared_size[3], griddims[3], blockdims[3]; 
    DataType* inputs[3] = {input_d, output_d, output_d + N * bs};
    DataType* outputs[3] = {output_d, output_d + N * bs, output_d};
    int kernel_launch_times = param[1];
    float gflops, elapsed_time;
    hipEvent_t fft_begin, fft_end;
    // printf("adasdas\n");
    for(int i = 0; i < kernel_launch_times; ++i){
        // threadblock_bs = min((kernel_launch_times < 2 && bs < threadblock_bs) ? bs : param[5 + i], param[5 + i]);
        threadblock_bs = param[5 + i];
        Ni = (1 << param[2 + i]); 
        WorkerFFTSize = param[8 + i]; 
        shared_size[i] = Ni * threadblock_bs * sizeof(DataType);
        griddims[i] = (N * bs) / (Ni * threadblock_bs);
        blockdims[i] = (Ni * threadblock_bs) / WorkerFFTSize;
        printf("kernel=%d: gridDim=%d, blockDim=%d, share_mem_size=%d\n", i, griddims[i], blockdims[i], shared_size[i]);
        hipFuncSetAttribute(reinterpret_cast<const void*>(turboFFTArr[logN][i]), hipFuncAttributeMaxDynamicSharedMemorySize, shared_size[i]);
    }
    
    hipEventCreate(&fft_begin);
    hipEventCreate(&fft_end);

    hipEventRecord(fft_begin);
    #pragma unroll
    for (int j = 0; j < ntest; ++j){
        #pragma unroll
        for(int i = 0; i < kernel_launch_times; ++i){
            turboFFTArr[logN][i]<<<griddims[i], blockdims[i], shared_size[i]>>>(inputs[i], outputs[i], twiddle_d, bs);
        }
        hipDeviceSynchronize();
    }
    hipEventRecord(fft_end);
    hipEventSynchronize(fft_begin);
    hipEventSynchronize(fft_end);
    hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);

    
    elapsed_time = elapsed_time / ntest;
    gflops = 5 * N * log2f(N) * bs / elapsed_time * 1000 / 1000000000.f;
    
    printf("turboFFT finished: T=%8.3fms, FLOPS=%8.3fGFLOPS\n", elapsed_time, gflops);
    
    checkCudaErrors(hipMemcpy((void*)output_turbofft, (void*)outputs[kernel_launch_times - 1], N * bs * sizeof(DataType), hipMemcpyDeviceToHost));
}


int main(int argc, char *argv[]){
    if (argc < 3) {
        std::cerr << "Usage: program_name N bs" << std::endl;
        return 1;
    }

    long long logN = std::atoi(argv[1]); // Convert first argument to integer
    long long N = 1 << logN; // Convert first argument to integer
    long long bs = std::atoi(argv[2]); // Convert second argument to integer
    bool if_profile = 1;
    bool if_verify = 0;
    if (argc >= 4) if_profile = std::atoi(argv[3]);
    if (argc >= 5) if_verify = std::atoi(argv[4]);
    
    DataType* input, *output_turbofft, *output_cufft;
    DataType* input_d, *output_d, *twiddle_d;
    int ntest = 10;

    std::vector<std::vector<long long int>> params;
    
    // std::ifstream file("../include/param/param.csv");
    // if (file.is_open()) {
    //     std::cout << "File opened successfully." << std::endl;
    //     // Perform file operations here
    // } else {
    //     std::cout << "Failed to open file." << std::endl;
    // }    
    // params = utils::load_parameters(file);
    std::string param_file_path = "../include/param/param.csv";
    params = utils::load_parameters(param_file_path);

    utils::initializeData<DataType>(input, input_d, output_d, output_turbofft, output_cufft, twiddle_d, N, bs + 3);


    // Verification
    if(if_verify){
        test_turbofft(input_d, output_d, output_turbofft, twiddle_d, params[logN], bs, 1);
        profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, N, bs, 1);
        
        utils::compareData<DataType>(output_turbofft, output_cufft, N * bs, 1e-5);
    }

    // Profiling
    if(if_profile){
        // test_turbofft(input_d, output_d, output_turbofft, twiddle_d, params[logN], bs, ntest);        
        
        profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, N, bs, ntest);
        test_turbofft(input_d, output_d, output_turbofft, twiddle_d, params[logN], bs, ntest);        

        
        // test_turbofft(input_d, output_d, output_turbofft, twiddle_d, params[logN], bs, ntest);
    }
    hipFree(input_d);
    hipFree(output_d);
    return 0;
}

// profiler::cufft::test_cufft_ft<DataType>(input_d, output_d, output_cufft, input_d + N * (bs + 2),
//                                          input_d + N * (bs + 1), output_d + N * (bs + 2),   N, bs, ntest);
// profiler::cufft::test_cufft_ft<DataType>(input_d, output_d, output_cufft, input_d + N * (bs + 2),
//                                          input_d + N * (bs + 1), output_d + N * (bs + 2),   N, bs, ntest);
