#include "hip/hip_runtime.h"
#include "include/turbofft.h"
// #include "include/code_gen/generated/fft_radix_2_logN_8_upload_0.cuh"
#include "include/code_gen/generated/fft_radix_2_logN_2_upload_0.cuh"
// #define DataType nv_bfloat162
#define DataType double2
// #define DataType float2
// #define DataType half2



void test_turbofft(DataType* input_d, DataType* output_d, DataType* output_turbofft, 
                    long long int N){
    dim3 gridDim(1, 1, 1); 
    dim3 blockDim(1, 1, 1);
    // turbofft::fft::thread::fft<DataType, turbofft::Tensor<DataType, 1, 2>><<<gridDim, blockDim>>>(input_d, output_d);
    // turbofft::fft::thread::fft<DataType><<<gridDim, blockDim>>>(input_d, output_d);
    long long int shared_size = N * sizeof(DataType);

    // hipFuncSetAttribute(reinterpret_cast<const void*>(turbofft::fft::thread::fft<DataType), 8>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_size);
    hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix_2_logN_2_dim_0), hipFuncAttributeMaxDynamicSharedMemorySize, shared_size);
    
    
    // turbofft::fft::thread::fft<DataType, 8><<<gridDim, blockDim, shared_size>>>(input_d, output_d);
    // turbofft::fft::thread::fft<DataType><<<gridDim, blockDim>>>(input_d, output_d);
    // fft_radix_2_logN_8_dim_0<<<gridDim, blockDim>>>(input_d, output_d);
    fft_radix_2_logN_2_dim_0<<<gridDim, blockDim>>>(input_d, output_d);
    hipDeviceSynchronize();
    printf("%d\n",  N * sizeof(DataType));
    checkCudaErrors(hipMemcpy((void*)output_turbofft, (void*)output_d, N * sizeof(DataType), hipMemcpyDeviceToHost));
}


int main(int argc, char *argv[]){
    DataType* input, *output_turbofft, *output_cufft;
    DataType* input_d, *output_d;
    long long int N = 1 << 2, bs = 1;
    int ntest = 1;

    if (argc < 2) bs = 1;
    else bs = std::atoi(argv[1]);
    printf("N=%d, bs=%d\n", N, bs);
    utils::initializeData<DataType>(input, input_d, output_d, output_turbofft, output_cufft, N, bs + 3);
    test_turbofft(input_d, output_d, output_turbofft, N);

    
    // profiler::cufft::test_cufft_ft<DataType>(input_d, output_d, output_cufft, input_d + N * (bs + 2),
    //                                          input_d + N * (bs + 1), output_d + N * (bs + 2),   N, bs, ntest);

    profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, N, bs, ntest);

    // profiler::cufft::test_cufft_ft<DataType>(input_d, output_d, output_cufft, input_d + N * (bs + 2),
    //                                          input_d + N * (bs + 1), output_d + N * (bs + 2),   N, bs, ntest);

    
    utils::compareData<DataType>(output_turbofft, output_cufft, N, 1e-5);
    // printData(output_turbofft, N);
    // printData(output_cufft, N);
    hipFree(input_d);
    hipFree(output_d);
    return 0;
}