#include "hip/hip_runtime.h"

#include "include/TurboFFT.h"
    
template <typename DataType>
void test_turbofft( DataType* input_d, DataType* output_d, DataType* output_turbofft,
                    DataType* twiddle_d, DataType* checksum, std::vector<long long int> param, 
                    long long int bs, int thread_bs, int ntest){
    long long int N = (1 << param[0]), threadblock_bs, Ni, WorkerFFTSize;
    long long int logN = param[0];
    long long int shared_size[3], griddims[3], blockdims[3]; 
    DataType* inputs[3] = {input_d, output_d, output_d + N * bs};
    DataType* outputs[3] = {output_d, output_d + N * bs, output_d};
    int kernel_launch_times = param[1];
    float gflops, elapsed_time, mem_bandwidth;
    hipEvent_t fft_begin, fft_end;
    
    hipblasHandle_t handle;      

    int M = 16;
    dim3 gridDim1((N + 255) / 256, bs / M, 1);
    
    TurboFFT_Kernel_Entry<DataType> entry;
    for(int i = 0; i < kernel_launch_times; ++i){
        threadblock_bs = param[5 + i];
        Ni = (1 << param[2 + i]); 
        WorkerFFTSize = param[8 + i]; 
        shared_size[i] = Ni * threadblock_bs * sizeof(DataType);
        
        blockdims[i] = (Ni * threadblock_bs) / WorkerFFTSize;
        long long int shared_per_SM = 160 * 1024;
        shared_per_SM = 128 * 1024;
        griddims[i] = min(108 * min((2048 / blockdims[i]), (shared_per_SM / shared_size[i])), 
                ((N * bs) + (Ni * threadblock_bs) - 1) / (Ni * threadblock_bs));
        
        griddims[i] = ((((N * bs) + (Ni * threadblock_bs) - 1) / (Ni * threadblock_bs))) / thread_bs;
    
        hipFuncAttributes attr;
        if(hipFuncSetAttribute(reinterpret_cast<const void*>(entry.turboFFTArr[logN][i]), hipFuncAttributeMaxDynamicSharedMemorySize, shared_size[i]))
        printf("Set DynamicSharedMem failed\n");
        if(hipFuncSetAttribute(reinterpret_cast<const void*>(entry.turboFFTArr[logN][i]), hipFuncAttributePreferredSharedMemoryCarveout, (shared_per_SM * 100) / (164 * 1024)))
        printf("Set smemCarveout failed\n");
        hipError_t get_attr_res = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(entry.turboFFTArr[logN][i] ));
        if(get_attr_res != 0)
        printf("get_attr_res = %d\n", get_attr_res);
    }
    
    hipEventCreate(&fft_begin);
    hipEventCreate(&fft_end);
    #pragma unroll
    for(int i = 0; i < kernel_launch_times; ++i){
        entry.turboFFTArr[logN][i]<<<griddims[i], blockdims[i], shared_size[i]>>>(inputs[i], outputs[i], twiddle_d, checksum, bs, thread_bs);
    }

    hipEventRecord(fft_begin);
    #pragma unroll
    for (int j = 0; j < ntest; ++j){
    
        #pragma unroll
        for(int i = 0; i < kernel_launch_times; ++i){
            entry.turboFFTArr[logN][i]<<<griddims[i], blockdims[i], shared_size[i]>>>(inputs[i], outputs[i], twiddle_d, checksum, bs, thread_bs);
            hipDeviceSynchronize();
        }
    
        hipDeviceSynchronize();
    }
    hipEventRecord(fft_end);
    hipEventSynchronize(fft_begin);
    hipEventSynchronize(fft_end);
    hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
    elapsed_time = elapsed_time / ntest;
    gflops = 5 * N * log2f(N) * bs / elapsed_time * 1000 / 1000000000.f;
    mem_bandwidth = (float)(N * bs * sizeof(DataType) * 2) / (elapsed_time) * 1000.f / 1000000000.f;
    printf("turboFFT, %d, %d, %8.3f, %8.3f, %8.3f\n",  (int)log2f(N),  (int)log2f(bs), elapsed_time, gflops, mem_bandwidth);
    
    checkCudaErrors(hipMemcpy((void*)output_turbofft, (void*)outputs[kernel_launch_times - 1], N * bs * sizeof(DataType), hipMemcpyDeviceToHost));
}


template <typename DataType>
void TurboFFT_main(ProgramConfig &config){


    DataType* input, *output_turbofft, *output_cufft;
    DataType* input_d, *output_d, *twiddle_d;
    int ntest = 10;

    std::vector<std::vector<long long int>> params;
    
    params = utils::load_parameters(config.param_file_path);

    DataType* checksum_d, *checksum_h;
    hipMalloc((void**)&checksum_d, sizeof(DataType) * 16384 * 2);
    checksum_h = (DataType*)calloc(16384 * 2, sizeof(DataType));
    DataType* dest = checksum_h;
    for(int i = 2; i <= (1 << 13); i *= 2){
        utils::getDFTMatrixChecksum(dest, i);
        dest += i;
    }
    // utils::printData<DataType>(checksum_h + 62, 64);
    hipMemcpy((void*)checksum_d, (void*)checksum_h, sizeof(DataType) * 16384 * 2, hipMemcpyHostToDevice);


    
    if(config.if_bench){
        // Verification
        utils::initializeData<DataType>(input, input_d, output_d, output_turbofft, output_cufft, twiddle_d, config.N, config.bs_end);

        if(config.if_verify){
            test_turbofft<DataType>(input_d, output_d, output_turbofft, twiddle_d, checksum_d, params[logN], config.bs, config.thread_bs, 1);
            profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, config.N, config.bs, 1);            
            utils::compareData<DataType>(output_turbofft, output_cufft, config.N * config.bs, 1e-4);
        }
        // Profiling
        if(if_profile){
            long long int bs_begin = config.bs;
            for(bs = bs_begin; bs <= config.bs_end; bs += config.bs_gap)
            profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, N, bs, ntest);
            
            for(bs = bs_begin; bs <= config.bs_end; bs += config.bs_gap)
            test_turbofft<DataType>(input_d, output_d, output_turbofft, twiddle_d, checksum_d, params[logN], config.bs, config.thread_bs, ntest);
        }
    }
    
    if(if_bench){
        utils::initializeData<DataType>(input, input_d, output_d, output_turbofft, output_cufft, twiddle_d, 1 << 25, 16 + 3);
        N = 1;
        for(logN = 1; logN <= 25; ++logN){
            N *= 2;
            bs = 1;
            // bs = bs << (28-logN);
            for(int i = 0; i < 29 - logN; i += 1){
                // profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, N, bs, ntest);
                test_turbofft<DataType>(input_d, output_d, output_turbofft, twiddle_d, checksum_d, params[logN], bs, config.thread_bs, ntest);
                bs *= 2;
                // break; 
            }
        }

   

    }
    hipFree(input_d);
    hipFree(output_d);
    hipFree(twiddle_d);
    free(input);
    free(output_cufft);
    free(output_turbofft);
}

int main(int argc, char *argv[]){
    ProgramConfig config;
    if (!config.parseCommandLine(argc, argv)) {
        return 0; // Early exit if help was requested or an error occurred
    }
    
    config.displayConfig();
    // Proceed with the rest of the program
    

    if(config.datatype == 0) {
        TurboFFT_main<float2>(config);
    }
    else {
        TurboFFT_main<double2>(config);
    }
    
    return 0;
}


    