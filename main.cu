#include "hip/hip_runtime.h"
#include "include/turbofft.h"
// #define DataType nv_bfloat162
// #define DataType double2
#define DataType float2
// #define DataType half2



void test_turbofft(DataType* input_d, DataType* output_d, DataType* output_turbofft, 
                    long long int N){
    dim3 gridDim(1, 1, 1); 
    dim3 blockDim(1, 32, 1);
    // turbofft::fft::thread::fft<DataType, turbofft::Tensor<DataType, 1, 2>><<<gridDim, blockDim>>>(input_d, output_d);
    // turbofft::fft::thread::fft<DataType><<<gridDim, blockDim>>>(input_d, output_d);
    long long int shared_size = (N * sizeof(DataType) / 16) * 17;

    hipFuncSetAttribute(reinterpret_cast<const void*>(turbofft::fft::thread::fft<DataType), 8>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_size);
    turbofft::fft::thread::fft<DataType, 8><<<gridDim, blockDim, shared_size>>>(input_d, output_d);
    // turbofft::fft::thread::fft<DataType><<<gridDim, blockDim>>>(input_d, output_d);
    hipDeviceSynchronize();
    printf("%d\n",  N * sizeof(DataType));
    checkCudaErrors(hipMemcpy((void*)output_turbofft, (void*)output_d, N * sizeof(DataType), hipMemcpyDeviceToHost));

}


int main(){
    DataType* input, *output_turbofft, *output_cufft;
    DataType* input_d, *output_d;
    long long int N = 2 << 10, bs = 2 << 10;
    int ntest = 10;

    utils::initializeData<DataType>(input, input_d, output_d, output_turbofft, output_cufft, N, bs);
    // test_turbofft(input_d, output_d, output_turbofft, N);

    profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, N, bs, ntest);

    profiler::cufft::test_cufft_ft<DataType>(input_d, output_d, output_cufft, input_d + N * (bs + 2),
                                             input_d + N * (bs + 1), output_d + N * (bs + 2),   N, bs + 1, ntest);

    // utils::compareData<DataType>(output_turbofft, output_cufft, N, 1e-5);
    // printData(output_turbofft, N);
    // printData(output_cufft, N);

    return 0;
}